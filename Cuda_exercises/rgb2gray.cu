#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

/* __global__ specifier tells the CUDA C++ compiler that this is a function 
	that runs on the GPU and can be called from CPU code. 
 */

__global__
void rgb2grayGPU_kernel(const uchar3* const dev_rgb_image,
                       unsigned char* const dev_gray_image,
                       int num_rows, int num_cols)
{
	uchar R, G, B;

	// Calculate the index of the pointers based on pixel location for each thread
    int x = blockIdx.x * blockDim.x + threadIdx.x; //x coordinate of pixel
    int y = blockIdx.y * blockDim.y + threadIdx.y; //y coordinate of pixel

    if ((x >= num_cols) && (y >= num_rows)) 
    {
        return;
    }

    int index = y*num_cols + x; //absolute pixel position
    uchar3 pixel = dev_rgb_image[index];

	B = pixel.x;
	G = pixel.y;
	R = pixel.z;

	dev_gray_image[index] = (uchar)(0.299*R + 0.587*G + 0.114*B);
}


Mat rgb2grayGPU(Mat rgb_image)
{
  	const int num_rows = rgb_image.rows;
  	const int num_cols = rgb_image.cols;
  	const int total_pixels = num_rows*num_cols;


	/* Adapt image format and copy to device */
	/*****************************************/

	uchar3 *host_rgb_image = (uchar3 *)rgb_image.ptr<unsigned char>(0);

	uchar3 *device_rgb_image;
	unsigned char *device_gray_image;

	hipMalloc(&device_rgb_image, sizeof(uchar3) * total_pixels);
	hipMalloc(&device_gray_image, sizeof(unsigned char) * total_pixels);

	hipMemset(device_gray_image, 0, total_pixels * sizeof(unsigned char));

	 //copy input array to the GPU
	hipMemcpy(device_rgb_image, host_rgb_image, sizeof(uchar3) * total_pixels, hipMemcpyHostToDevice);


	/* Execute kernel in GPU */
	/*************************/

	const int thread = 16;
	const dim3 blockSize( thread, thread);
  	const dim3 gridSize( ceil(num_cols/(float)thread) + 1, ceil(num_rows/(float)thread) + 1);

	rgb2grayGPU_kernel<<<gridSize, blockSize>>>(device_rgb_image, device_gray_image, num_rows, num_cols);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();


	/* Gather image from device and free memory */
	/********************************************/
	Mat host_gray_image(num_rows, num_cols, CV_8UC1, Scalar(0));
	hipMemcpy(host_gray_image.ptr<unsigned char>(0), device_gray_image, sizeof(unsigned char) * total_pixels, hipMemcpyDeviceToHost);

	hipFree(device_rgb_image);
	hipFree(device_gray_image);


	return host_gray_image;
}


